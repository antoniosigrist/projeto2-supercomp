#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
using namespace std;
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include <chrono>
using namespace std::chrono;

#define RND (hiprand_uniform(&local_rand_state))
#define seed 1000

 __device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); 
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(seed, 0, 0, rand_state); //cria uma seed na thread 0 que vai garantir mesma seed para todas as threads
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) 
        return;
    int pixel_index = j*max_x + i;
    
    hiprand_init(seed, pixel_index, 0, &rand_state[pixel_index]); //repassa o mesmo rand_state para os threads dos blocos utilizados na compilacao para garantir consistencia. 
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) 
        return; // garante que nao vai rodar alem dos tamanho definido no kernel
    int pixel_index = j*max_x + i; // calcula a posicao do pixel no kernel
    hiprandState local_rand_state = rand_state[pixel_index]; 
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);

        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col; //coloca o resultado em fb para ser acessado do host ao final do código
}


__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2, new lambertian(vec3(hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state), RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2, new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(25,12,13);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) { //deleta espaco alocado para cada esfera
    for(int i=0; i < 22*22+1+3; i++) { 
        delete ((sphere *)d_list[i])->mat_ptr; 
        delete d_list[i];
    }
    delete *d_world; 
    delete *d_camera;
}

int main() {

    ofstream myfile;
    myfile.open ("tempo.txt");

    int num_testes = 30;

    int prop;
    
    for(int k = 1;k<num_testes;k++) {

    prop = k;

    int nx = (int) 1200/prop;
    int ny = (int) 800/prop;
    int ns = 10;
    int tx = 8;
    int ty = 8;


    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3); //aloca tamanho do vetor fb para cada pixel caber um vec3


    vec3 *fb;
    hipMallocManaged((void **)&fb, fb_size); //aloca lista do tamanho do numero de pixels da imagem. Cudamallocmanage "copia" o mesmo endereco de memoria para CPU e GPU.

  
    hiprandState *d_rand_state;
    hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState));
    hiprandState *d_rand_state2;
    hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState));

    rand_init<<<1,1>>>(d_rand_state2); // inicializa kernel que cria seed no bloco 0 thread 0

    //hipDeviceSynchronize();  - Utilizado apenas para debug, não é necessário dado que threads são assincrona entre si, mas sequenciais entre elas

    hitable **d_list;
    int num_hitables = 489; //tem que ser maior ou igual a 488 que é o numero de bolinhas criadas :)
    hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)); //aloca hitables
    hitable **d_world;
    hipMalloc((void **)&d_world, sizeof(hitable *));
    camera **d_camera;
    hipMalloc((void **)&d_camera, sizeof(camera *));

    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2); //cria mundo randomico utilizando o estado incial criado por rand_init no bloco 0 thread 0


    clock_t start, stop;
    start = clock();

    dim3 blocks(nx/tx+1,ny/ty+1); //define o numero de blocos (tx e ty são multiplos de 8 já que a arquitetura de 8x8 threads, garantindo que cada bloco faca um numero pareceido de processamento)
    dim3 threads(tx,ty);

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state); //cria o kernel de tamanho block x threads. 


    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state); // renderiza a imagem no tamanho do bloco e threads estabelicidos, garantindo o mesmo cenário para todas as threads (maior parte do processamento está aqui)


    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;


    myfile << "Tamanho da Imagens x Tempo de Execução: ";
    myfile << "\n";
    myfile << "Tamanho da Imagem: "<< nx <<" x " << ny << " - Tempo de Execução: " << timer_seconds << "," << "\n"; //escreve tempo de execucao e tamanho da imagem da imagem rodada.

    // Como estamos realizando diversos testes de tamanhos de imagem diferente, desejamos que apenas uma imagem seja criada para podermos analisar a qualidade
    if(k==2){ //devolve apenas os pixels do tamanho de prop==2

        hipDeviceSynchronize(); //garante que processamento já acabou para acessar dados de fb

        std::cout << "P3\n" << nx << " " << ny << "\n255\n";
        for (int j = ny-1; j >= 0; j--) {
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = j*nx + i;
                int ir = int(255.99*fb[pixel_index].r()); //pega cor vermelha de fb
                int ig = int(255.99*fb[pixel_index].g()); // pega cor verde de fb
                int ib = int(255.99*fb[pixel_index].b());//pega cor azul de fb
                std::cout << ir << " " << ig << " " << ib << "\n";
            }
        }
    }

    // limpando a memoria alocada

    free_world<<<1,1>>>(d_list,d_world,d_camera);

    hipFree(d_list);
    hipFree(d_rand_state);
    hipFree(fb);
    hipFree(d_camera);
    hipFree(d_world);
    

    hipDeviceReset();

    }

    myfile.close(); //fecha arquivo de escrita
}









